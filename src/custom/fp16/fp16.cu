#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu_fp16.h"

#define cudaErrChk(stmt) \
  { cudaAssert((stmt), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t error,
                       const char* file,
                       int line,
                       bool abort = true) {
  if (error != hipSuccess) {
    std::cerr << "CUDA error: "
              << hipGetErrorString(error) << ' ' << file << ':' << line << std::endl;
    if (abort) {
      exit(error);
    }
  }
}

// Stream configurations
//#define USE_STREAM  // Use multi-stream to accelerate transfers
#define N_STREAMS 16

// Allocate maximal possible kernel size and reuse it between op1/2
#define M_MAX 16
#define C_MAX 4
#define KERNEL_WIDTH 7
__constant__ float kernel[M_MAX * C_MAX * KERNEL_WIDTH * KERNEL_WIDTH];
__constant__ int3 conv_lut[C_MAX * KERNEL_WIDTH * KERNEL_WIDTH];

// Tile configurations
#define TILE_WIDTH 8
#define PADDED_TILE_WIDTH (TILE_WIDTH + KERNEL_WIDTH - 1)

// NOTE:
//  B_BATCH * [
//    (HW_TILE_WIDTH + M_TILE_WIDTH) * CKK_TILE_WIDTH
//    + TILE_WIDTH * TILE_WIDTH
//  ]
#define HW_TILE_WIDTH (TILE_WIDTH * TILE_WIDTH)
#define M_TILE_WIDTH 4
#define CKK_TILE_WIDTH 16

// Block size along the B (batch) dimension
#define B_BATCH 4

__global__ void conv_as_gemm(float* __restrict__ y,
                             const float* __restrict__ x,
                             const int B,
                             const int M,
                             const int C,
                             const int H,
                             const int W,
                             const int K) {
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

  // Alias for height/width axis
  const int t_hw = threadIdx.x, b_hw = HW_TILE_WIDTH;
  // Alias for output channels
  const int t_m = threadIdx.y, b_m = M_TILE_WIDTH;
  const int m = blockIdx.y * b_m + t_m;
  // Alias for batch axis
  const int t_b = threadIdx.z;
  const int b = blockIdx.z * blockDim.z + t_b;

  // Y = (H W) * (K^2 C)
  /*
      c - input feature map
  ho/wo - output height/width
  hi/wi - input height/width
  hk/wk - convolution loop height/width
  */

  /*
   y.shape = (B, M, H_out, W_out)
   t.shape = (b, 0/1, H_out, W_out, C, K, K)
   k.shape = (M, C * K * K)
  */
#define y1d(i_hw)               \
  y[(b) * (M * H_out * W_out) + \
    (m) * (H_out * W_out) +     \
    (i_hw)]
#define x3d(i_c, i_ih, i_iw) \
  x[(b) * (C * H * W) +      \
    (i_c) * (H * W) +        \
    (i_ih) * (W) +           \
    (i_iw)]
#define k1d(i_ckk)           \
  kernel[(m) * (C * K * K) + \
         (i_ckk)]

  __shared__ half tile_xc[B_BATCH][HW_TILE_WIDTH][CKK_TILE_WIDTH];
  __shared__ half tile_kt[B_BATCH][M_TILE_WIDTH][CKK_TILE_WIDTH];
  __shared__ half tile_x[B_BATCH][C_MAX][PADDED_TILE_WIDTH][PADDED_TILE_WIDTH];

#define t2d_kt(i_m, i_ckk) \
  tile_kt[t_b][i_m][i_ckk]
#define t2d_xc(i_ckk, i_hw) \
  tile_xc[t_b][i_hw][i_ckk]

  const int n_hw = H_out * W_out;

  // Aggregate all threads, we will reassign their index
  // NOTE: HW_TILE_WIDTH * M_TILE_WIDTH = TILE_WIDTH * TILE_WIDTH

  const int ib_hw = blockIdx.x;
  const int nb_w = (W_out + (TILE_WIDTH - 1)) / TILE_WIDTH;
  // .. Block index in 2D grid
  const int ib_w = ib_hw % nb_w;
  const int ib_h = ib_hw / nb_w;
  // .. Thread index in 2D block
  const int t_w = t_hw % TILE_WIDTH;
  const int t_h = t_hw / TILE_WIDTH;

  const int dst_w = ib_w * TILE_WIDTH + t_w;
  const int dst_h = ib_h * TILE_WIDTH + t_h;
  const int dst_hw = dst_h * W_out + dst_w;

  // Calculate number of subtiles
  const int n_kernel = C * K * K;
  const int n_tiles = (n_kernel + (CKK_TILE_WIDTH - 1)) / CKK_TILE_WIDTH;

  int t_pw, t_ph, src_w, src_h;
  if ((b < B)) {
    // Pre-load to shared memory, need to loop multiple time, PW^2 / W^2
    for (int c = 0; c < C; c++) {
      for (int dst = t_m * HW_TILE_WIDTH + t_hw;
           dst < PADDED_TILE_WIDTH * PADDED_TILE_WIDTH;
           dst += TILE_WIDTH * TILE_WIDTH) {
        // 2D block index in a padded tile
        t_pw = dst % PADDED_TILE_WIDTH;
        t_ph = dst / PADDED_TILE_WIDTH;
        // 3D index in global array, simply subtract the pad size
        src_w = ib_w * TILE_WIDTH + t_pw;
        src_h = ib_h * TILE_WIDTH + t_ph;

        if ((src_w < W) && (src_h < H)) {
          tile_x[t_b][c][t_ph][t_pw] = __float2half(x3d(c, src_h, src_w));
        } else {
          tile_x[t_b][c][t_ph][t_pw] = __float2half(0.0f);
        }
      }
    }
    __syncthreads();

    half2 acc2 = __half2half2(__float2half(0.0f));
    for (int n = 0; n < n_tiles; n++) {
      // Save sub-tile of xc and kernel to smem
      for (int i_ckk = t_m; i_ckk < CKK_TILE_WIDTH; i_ckk += M_TILE_WIDTH) {
        int dst_ckk = n * CKK_TILE_WIDTH + i_ckk;
        if ((dst_w < W_out) && (dst_h < H_out) && (dst_ckk < n_kernel)) {
          // Do input matrix unroll on-the-fly
          int3 lut = conv_lut[dst_ckk];
          const int q = lut.x, p = lut.y, c = lut.z;

          // Load from global memory to tiled memory as unrolled column matrix
          tile_xc[t_b][t_hw][i_ckk] = tile_x[t_b][c][t_h + p][t_w + q];
        } else {
          tile_xc[t_b][t_hw][i_ckk] = __float2half(0.0f);
        }
      }

      for (int i_ckk = t_hw; i_ckk < CKK_TILE_WIDTH; i_ckk += HW_TILE_WIDTH) {
        const int dst_ckk = n * CKK_TILE_WIDTH + i_ckk;
        if ((m < M) && (dst_ckk < n_kernel)) {
          t2d_kt(t_m, i_ckk) = __float2half(k1d(dst_ckk));
        } else {
          t2d_kt(t_m, i_ckk) = __float2half(0.0f);
        }
      }

      /*
      dst_ckk = n * TILE_WIDTH + t_m;
      if ((dst_hw < n_hw) && (dst_ckk < n_kernel)) {
        // Do input matrix unroll on-the-fly
        int3 lut = conv_lut[dst_ckk];
        const int q = lut.x, p = lut.y, c = lut.z;
        t2d_xc(t_m, t_hw) = x3d(c, dst_y + p, dst_x + q);
      } else {
        t2d_xc(t_m, t_hw) = 0.0;
      }

      dst_ckk = n * TILE_WIDTH + t_hw;
      if ((m < M) && (dst_ckk < n_kernel)) {
        t2d_kt(t_m, t_hw) = k1d(dst_ckk);
      } else {
        t2d_kt(t_m, t_hw) = 0.0;
      }
      */

      __syncthreads();

      // C_ij = A_ik * B_kj ===> C_ij^T = B_kj^T * A_ik^t
      half2* tile_kt2 = (half2*)&tile_kt[t_b][t_m][0];
      half2* tile_xc2 = (half2*)&tile_xc[t_b][t_hw][0];
#pragma unroll
      for (int k = 0; k < CKK_TILE_WIDTH / 2; k++) {
        acc2 = __hfma2(tile_kt2[k], tile_xc2[k], acc2);
      }
      __syncthreads();
    }

    if ((m < M) && (dst_w < W_out) && (dst_h < H_out)) {
      float2 acc = __half22float2(acc2);
      y1d(dst_hw) = acc.x + acc.y;
    }
  }

#undef t2d_xc
#undef t2d_kt

#undef y1d
#undef x3d
#undef t2d
#undef k1d
}

__host__ void GPUFP16Interface::conv_forward_gpu_prolog(const float* host_y,
                                                    const float* host_x,
                                                    const float* host_k,
                                                    float** device_y_ptr,
                                                    float** device_x_ptr,
                                                    float** device_k_ptr,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
  std::cout << "*** constant mem + tiled gemm" << std::endl;
  printf("*** (B=%d, M=%d, C=%d, H=%d, W=%d, K=%d)\n", B, M, C, H, W, K);

  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;
  printf("*** (H_out=%d, W_out=%d)\n", H_out, W_out);

  // Calculate needed bytes for original input
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);
  const size_t bytes_x = (B * C * H * W) * sizeof(float);
  const size_t bytes_k = (M * C * K * K) * sizeof(float);

#ifdef USE_STREAM
  // Pass through host pointers
  *device_y_ptr = (float*)host_y;
  *device_x_ptr = (float*)host_x;

  // Mark them as pinned memory for asynchronous transfer
  cudaErrChk(hipHostRegister(*device_y_ptr, bytes_y, hipHostRegisterPortable));
  cudaErrChk(hipHostRegister(*device_x_ptr, bytes_x, hipHostRegisterPortable));
#else
  // Allocate memory on device
  cudaErrChk(hipMalloc(device_y_ptr, bytes_y));
  cudaErrChk(hipMalloc(device_x_ptr, bytes_x));

  // Copy input data to device
  cudaErrChk(hipMemcpy(*device_x_ptr, host_x, bytes_x, hipMemcpyHostToDevice));
#endif

  // Copy kernel weights
  cudaErrChk(hipMemcpyToSymbol(HIP_SYMBOL(kernel), host_k, bytes_k));

  // Calculate lookup table
  int3 host_conv_lut[C * K * K];
  for (int i = 0, c = 0; c < C; c++) {
    for (int p = 0; p < K; p++) {
      for (int q = 0; q < K; q++, i++) {
        host_conv_lut[i] = make_int3(q, p, c);
      }
    }
  }
  const size_t bytes_conv_lut = (C * K * K) * sizeof(int3);
  cudaErrChk(hipMemcpyToSymbol(HIP_SYMBOL(conv_lut), host_conv_lut, bytes_conv_lut));
}

__host__ void GPUFP16Interface::conv_forward_gpu(float* device_y,
                                             const float* device_x,
                                             const float* device_k,  // unused
                                             const int B0,
                                             const int M,
                                             const int C,
                                             const int H,
                                             const int W,
                                             const int K) {
  // Estimat output dimension
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;

/*** Prolog BEGIN ***/
#ifdef USE_STREAM
  // Create streams
  hipStream_t stream[N_STREAMS];
  for (int i = 0; i < N_STREAMS; i++) {
    cudaErrChk(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
  }

  // We pass through host pointers from the prolog function
  const float* host_x = device_x;
  float* host_y = device_y;

  // Calculate total elements and bytes
  const int n_y = B0 * M * H_out * W_out;
  const int n_x = B0 * C * H * W;
  const size_t bytes_y = n_y * sizeof(float);
  const size_t bytes_x = n_x * sizeof(float);
  // Calculate partial elements and bytes per stream
  const int B = ceil((float)B0 / N_STREAMS);
  const int n_y_stream = B * M * H_out * W_out;
  const int n_x_stream = B * C * H * W;

  cudaErrChk(hipMalloc(&device_y, bytes_y));
  cudaErrChk(hipMalloc(&device_x, bytes_x));

  // Copy over the relevant data structures to the GPU
  for (int i = 0; i < N_STREAMS; i++) {
    size_t offset = i * n_x_stream;
    size_t bytes = n_x_stream * sizeof(float);
    if (offset + n_x_stream > n_x) {
      // Last stream does not need to copy that much
      bytes = (n_x - offset) * sizeof(float);
    }
    cudaErrChk(hipMemcpyAsync((void*)&device_x[offset], (void*)&host_x[offset], bytes,
                               hipMemcpyHostToDevice, stream[i]));
  }
#else   // USE_STREAM
  // Send the entire batch
  const int B = B0;
#endif  // USE_STREAM
  /*** Prolog END ***/

  /*** Kernel call BEGIN ***/
  // Calculate launch size
  dim3 block(HW_TILE_WIDTH, M_TILE_WIDTH, B_BATCH);
  dim3 grid(ceil((float)H_out / TILE_WIDTH) * ceil((float)W_out / TILE_WIDTH),
            ceil((float)M / M_TILE_WIDTH),
            ceil((float)B / B_BATCH));
  printf("*** grid=(x=%d, y=%d, z=%d), block=(x=%d, y=%d, z=%d)\n",
         grid.x, grid.y, grid.z, block.x, block.y, block.z);

  // Determine shared memory size
  size_t smem_size =
      B_BATCH * ((HW_TILE_WIDTH + M_TILE_WIDTH) * CKK_TILE_WIDTH + C_MAX * PADDED_TILE_WIDTH * PADDED_TILE_WIDTH) * sizeof(float);
  std::cout << "*** estimated smem.size=" << smem_size / 1024 << "KiB" << std::endl;
  smem_size = 0;  // DEBUG

// Call the kernel
#ifdef USE_STREAM
  for (int i = 0; i < N_STREAMS; i++) {
    size_t offset_y = i * n_y_stream;
    size_t offset_x = i * n_x_stream;
    conv_as_gemm<<<grid, block, smem_size, stream[i]>>>(
        &device_y[offset_y], &device_x[offset_x],
        B, M, C, H, W, K);
  }
#else   // USE_STREAM
  conv_as_gemm<<<grid, block, smem_size>>>(device_y, device_x,
                                           B, M, C, H, W, K);
#endif  // USE_STREAM
  /*** Kernel call END ***/

  /*** Epilog BEGIN ***/
#ifdef USE_STREAM
  // Copy back data to host
  for (int i = 0; i < N_STREAMS; i++) {
    size_t offset = i * n_y_stream;
    size_t bytes = n_y_stream * sizeof(float);
    if (offset + n_y_stream > n_y) {
      // Last stream does not need to copy that much
      bytes = (n_y - offset) * sizeof(float);
    }
    cudaErrChk(hipMemcpyAsync(&host_y[offset], &device_y[offset], bytes,
                               hipMemcpyDeviceToHost, stream[i]));
  }

  // Destory streams
  for (int i = 0; i < N_STREAMS; i++) {
    cudaErrChk(hipStreamDestroy(stream[i]));
  }
#else   // USE_STREAM
  // nop
#endif  // USE_STREAM
  /*** Epilog END ***/

  // We directly wait for the single kernel to end
  cudaErrChk(hipDeviceSynchronize());
}

__host__ void GPUFP16Interface::conv_forward_gpu_epilog(float* host_y,
                                                    float* device_y,
                                                    float* device_x,
                                                    float* device_k,
                                                    const int B,
                                                    const int M,
                                                    const int C,
                                                    const int H,
                                                    const int W,
                                                    const int K) {
#ifdef USE_STREAM
  // Data is already write back to host earlier, safe to clean up now

  // Release pinned memory
  cudaErrChk(hipHostUnregister(device_y));
  cudaErrChk(hipHostUnregister(device_x));
#else   // USE_STREAM
  const int H_out = H - K + 1;
  const int W_out = W - K + 1;
  const size_t bytes_y = (B * M * H_out * W_out) * sizeof(float);

  // Copy output back to host
  cudaErrChk(hipMemcpy(host_y, device_y, bytes_y, hipMemcpyDeviceToHost));

  // Free device memory
  cudaErrChk(hipFree(device_y));
  cudaErrChk(hipFree(device_x));
#endif  // USE_STREAM
}

__host__ void GPUFP16Interface::get_device_properties() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
    std::cout << "Computational capabilities: "
              << deviceProp.major << "." << deviceProp.minor << std::endl;
    std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem
              << std::endl;
    std::cout << "Max Constant memory size: " << deviceProp.totalConstMem
              << std::endl;
    std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock
              << std::endl;
    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock
              << std::endl;
    std::cout << "Max block dimensions: "
              << deviceProp.maxThreadsDim[0] << " x, "
              << deviceProp.maxThreadsDim[1] << " y, "
              << deviceProp.maxThreadsDim[2] << " z" << std::endl;
    std::cout << "Max grid dimensions: "
              << deviceProp.maxGridSize[0] << " x, "
              << deviceProp.maxGridSize[1] << " y, "
              << deviceProp.maxGridSize[2] << " z" << std::endl;
    std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
  }
}