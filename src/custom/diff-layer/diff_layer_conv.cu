#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

using namespace std;

#include "gpu_diff_layer.h"
#define TILE_WIDTH1 16
#define TILE_WIDTH2 24
__constant__ float Mask[6000];

using namespace std;

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

__host__ void GPUDiffLayerSizeInterface::get_device_properties() {
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");
}

__global__ void conv_forward_kernel1(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) Mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    // int numblock_eachcolumn = (H_out - 1)/TILE_WIDTH + 1;
    int numblock_eachrow = (W_out - 1)/TILE_WIDTH1 + 1;
    int w_out = TILE_WIDTH1 * (bz % numblock_eachrow) + tx;
    int h_out = TILE_WIDTH1 * (bz/numblock_eachrow) + ty;
    int b_out = bx;
    int m_out = by;

    if (h_out < H_out && w_out < W_out)
    {
        float result = 0;
        for (int c = 0; c < C; c++)
        {
            for (int p = 0; p < K; p++)
            {
                for (int q = 0; q < K; q++)
                {
                    result += x4d(b_out, c, h_out + p, w_out + q) * k4d(m_out, c, p, q);
                }
            }
        }
        y4d(b_out, m_out, h_out, w_out) = result;
    }
    





#undef y4d
#undef x4d
#undef k4d
}

__global__ void conv_forward_kernel2(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) Mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    // int numblock_eachcolumn = (H_out - 1)/TILE_WIDTH + 1;
    int numblock_eachrow = (W_out - 1)/TILE_WIDTH2 + 1;
    int w_out = TILE_WIDTH2 * (bz % numblock_eachrow) + tx;
    int h_out = TILE_WIDTH2 * (bz/numblock_eachrow) + ty;
    int b_out = bx;
    int m_out = by;

    if (h_out < H_out && w_out < W_out)
    {
        float result = 0;
        for (int c = 0; c < C; c++)
        {
            for (int p = 0; p < K; p++)
            {
                for (int q = 0; q < K; q++)
                {
                    result += x4d(b_out, c, h_out + p, w_out + q) * k4d(m_out, c, p, q);
                }
            }
        }
        y4d(b_out, m_out, h_out, w_out) = result;
    }
    





#undef y4d
#undef x4d
#undef k4d
}

	
__host__ void GPUDiffLayerSizeInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{

    CHECK(hipMalloc((void **) device_y_ptr, (B * M * (H - K + 1) * (W - K + 1))*sizeof(float)));
    CHECK(hipMalloc((void **) device_x_ptr, (B * C * H * W)*sizeof(float)));
    CHECK(hipMemcpy(*device_x_ptr, host_x, (B * C * H * W)*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(Mask), host_k, (M * C * K * K)*sizeof(float)));

}


__host__ void GPUDiffLayerSizeInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    if(M/24){
        dim3 dimBlock(TILE_WIDTH2, TILE_WIDTH2, 1);
        dim3 dimGrid(B, M, ceil((float)(H - K + 1)/TILE_WIDTH2)*ceil((float)(W - K + 1)/TILE_WIDTH2));
        conv_forward_kernel2<<<dimGrid, dimBlock>>>(device_y, device_x, device_k, B, M, C, H, W, K);
    }else{
        dim3 dimGrid(B, M, ceil((float)(H - K + 1)/TILE_WIDTH1)*ceil((float)(W - K + 1)/TILE_WIDTH1));
        dim3 dimBlock(TILE_WIDTH1, TILE_WIDTH1, 1);
        conv_forward_kernel1<<<dimGrid, dimBlock>>>(device_y, device_x, device_k, B, M, C, H, W, K);
    }
}


__host__ void GPUDiffLayerSizeInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    CHECK(hipMemcpy(host_y, device_y, (B * M * (H - K + 1) * (W - K + 1))*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(device_y));
    CHECK(hipFree(device_x));
}

